#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <cfloat>

#include "Core/AABB.h"
#include "Core/Ray.h"

using namespace Varjo;

AABB::AABB()
{
	min.x = min.y = min.z = FLT_MAX;
	max.x = max.y = max.z = -FLT_MAX;
}

AABB AABB::createFromMinMax(const Vector3& min_, const Vector3& max_)
{
	AABB aabb;

	aabb.min = min_;
	aabb.max = max_;

	return aabb;
}

AABB AABB::createFromCenterExtent(const Vector3& center, const Vector3& extent)
{
	AABB aabb;

	aabb.min = center - extent / 2.0f;
	aabb.max = center + extent / 2.0f;

	return aabb;
}

AABB AABB::createFromVertices(const Vector3& v0, const Vector3& v1, const Vector3& v2)
{
	Vector3 min_;

	min_.x = MIN(v0.x, MIN(v1.x, v2.x));
	min_.y = MIN(v0.y, MIN(v1.y, v2.y));
	min_.z = MIN(v0.z, MIN(v1.z, v2.z));

	Vector3 max_;

	max_.x = MAX(v0.x, MAX(v1.x, v2.x));
	max_.y = MAX(v0.y, MAX(v1.y, v2.y));
	max_.z = MAX(v0.z, MAX(v1.z, v2.z));

	return AABB::createFromMinMax(min_, max_);
}

// http://tavianator.com/fast-branchless-raybounding-box-intersections-part-2-nans/
CUDA_CALLABLE bool AABB::intersects(const Ray& ray) const
{
	float tmin = 0;
	float tmax = 0;
	float tymin = 0;
	float tymax = 0;

	if (tmin > tymax || tymin > tmax)
		return false;

	if (tymin > tmin)
		tmin = tymin;

	if (tymax < tmax)
		tmax = tymax;

	float tzmin = 0;
	float tzmax = 0;

	if (tmin > tzmax || tzmin > tmax)
		return false;

	if (tzmin > tmin)
		tmin = tzmin;

	if (tzmax < tmax)
		tmax = tzmax;

	return (tmin < ray.maxDistance) && (tmax > ray.minDistance);
}

void AABB::expand(const AABB& other)
{
	if (other.min.x < min.x)
		min.x = other.min.x;

	if (other.min.y < min.y)
		min.y = other.min.y;

	if (other.min.z < min.z)
		min.z = other.min.z;

	if (other.max.x > max.x)
		max.x = other.max.x;

	if (other.max.y > max.y)
		max.y = other.max.y;

	if (other.max.z > max.z)
		max.z = other.max.z;
}

Vector3 AABB::getCenter() const
{
	return (min + max) * 0.5;
}

Vector3 AABB::getExtent() const
{
	return max - min;
}

float AABB::getSurfaceArea() const
{
	Vector3 extent = getExtent();
	return 2.0f * (extent.x * extent.y + extent.z * extent.y + extent.x * extent.z);
}
