﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Core/Ray.h"

using namespace Varjo;

CUDA_CALLABLE void Ray::precalculate()
{
	inverseDirection = direction.inversed();

	directionIsNegative[0] = direction.x < 0.0f;
	directionIsNegative[1] = direction.y < 0.0f;
	directionIsNegative[2] = direction.z < 0.0f;
}
