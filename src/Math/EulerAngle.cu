#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Math/EulerAngle.h"
#include "Math/Vector3.h"
#include "Math/MathUtils.h"

using namespace Varjo;

CUDA_CALLABLE EulerAngle::EulerAngle(float pitch_, float yaw_, float roll_) : pitch(pitch_), yaw(yaw_), roll(roll_)
{
}

namespace Varjo
{
	CUDA_CALLABLE EulerAngle operator+(const EulerAngle& e1, const EulerAngle& e2)
	{
		return EulerAngle(e1.pitch + e2.pitch, e1.yaw + e2.yaw, e1.roll + e2.roll);
	}

	CUDA_CALLABLE EulerAngle operator-(const EulerAngle& e1, const EulerAngle& e2)
	{
		return EulerAngle(e1.pitch - e2.pitch, e1.yaw - e2.yaw, e1.roll - e2.roll);
	}

	CUDA_CALLABLE EulerAngle operator*(const EulerAngle& e, float s)
	{
		return EulerAngle(e.pitch * s, e.yaw * s, e.roll * s);
	}

	CUDA_CALLABLE EulerAngle operator*(float s, const EulerAngle& e)
	{
		return EulerAngle(e.pitch * s, e.yaw * s, e.roll * s);
	}

	CUDA_CALLABLE EulerAngle operator-(const EulerAngle& e)
	{
		return EulerAngle(-e.pitch, -e.yaw, -e.roll);
	}
}

CUDA_CALLABLE EulerAngle& EulerAngle::operator+=(const EulerAngle& e)
{
	*this = *this + e;
	return *this;
}

CUDA_CALLABLE EulerAngle& EulerAngle::operator-=(const EulerAngle& e)
{
	*this = *this - e;
	return *this;
}

CUDA_CALLABLE void EulerAngle::clampPitch()
{
	if (pitch > 89.0f)
		pitch = 89.0f;

	if (pitch < -89.0f)
		pitch = -89.0f;
}

CUDA_CALLABLE void EulerAngle::normalize()
{
	while (std::abs(pitch) > 180.0f)
		pitch += (pitch > 0.0f) ? -360.0f : 360.0f;

	while (std::abs(yaw) > 180.0f)
		yaw += (yaw > 0.0f) ? -360.0f : 360.0f;

	while (std::abs(roll) > 180.0f)
		roll += (roll > 0.0f) ? -360.0f : 360.0f;
}

CUDA_CALLABLE Vector3 EulerAngle::getDirection() const
{
	Vector3 result;

	// is [0 0 -1] when angles are zero
	result.x = -std::sin(MathUtils::degToRad(yaw)) * std::cos(MathUtils::degToRad(pitch));
	result.y = std::sin(MathUtils::degToRad(pitch));
	result.z = -std::cos(MathUtils::degToRad(yaw)) * std::cos(MathUtils::degToRad(pitch));

	return result.normalized();
}

CUDA_CALLABLE EulerAngle EulerAngle::lerp(const EulerAngle& e1, const EulerAngle& e2, float t)
{
	EulerAngle result;
	float oneMinusT = 1.0f - t;

	result.pitch = e1.pitch * oneMinusT + e2.pitch * t;
	result.yaw = e1.yaw * oneMinusT + e2.yaw * t;
	result.roll = e1.roll * oneMinusT + e2.roll * t;

	return result;
}
