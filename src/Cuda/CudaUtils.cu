#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <hip/hip_runtime_api.h>
#include <>

#include "tinyformat/tinyformat.h"

#include "Cuda/CudaUtils.h"
#include "Utils/App.h"
#include "Utils/StringUtils.h"

using namespace Varjo;

void CudaUtils::checkError(hipError_t code, const std::string& message)
{
	if (code != hipSuccess)
		throw std::runtime_error(tfm::format("Cuda error: %s: %s", message, hipGetErrorString(code)));
}

void CudaUtils::initCuda()
{
	Log& log = App::getLog();
	Settings& settings = App::getSettings();

	checkError(hipSetDevice(settings.general.cudaDeviceNumber), "Could not set CUDA device");

	int deviceNumber;
	checkError(hipGetDevice(&deviceNumber), "Could not get CUDA device number");

	int deviceCount;
	checkError(hipGetDeviceCount(&deviceCount), "Could not get CUDA device count");

	hipDeviceProp_t deviceProp;
	checkError(hipGetDeviceProperties(&deviceProp, settings.general.cudaDeviceNumber), "Could not get CUDA device properties");

	int runtimeVersion;
	checkError(hipRuntimeGetVersion(&runtimeVersion), "Could not get CUDA runtime version");

	int driverVersion;
	checkError(hipDriverGetVersion(&driverVersion), "Could not get CUDA driver version");

	log.logInfo("CUDA Runtime version: %d | Driver version: %d", runtimeVersion, driverVersion);
	log.logInfo("CUDA device: %s (device number: %d, device count: %d)", deviceProp.name, deviceNumber, deviceCount);
	log.logInfo("CUDA Compute capability: %d.%d | Total memory: %s", deviceProp.major, deviceProp.minor, StringUtils::humanizeNumber(double(deviceProp.totalGlobalMem), true));
}

void CudaUtils::calculateDimensions(const void* kernel, const char* name, uint32_t length, int& blockSize, int& gridSize)
{
	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, length);

	gridSize = (length + blockSize - 1) / blockSize;

	App::getLog().logInfo("Kernel (%s) block size: %d | grid size: %d", name, blockSize, gridSize);
}

void CudaUtils::calculateDimensions2D(const void* kernel, const char* name, uint32_t width, uint32_t height, dim3& blockSize, dim3& gridSize)
{
	int tempBlockSize;
	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &tempBlockSize, kernel, 0, width * height);

	blockSize.x = 32;
	blockSize.y = tempBlockSize / 32;

	if (blockSize.y == 0)
		blockSize.y = 1;

	gridSize.x = (width + blockSize.x - 1) / blockSize.x;
	gridSize.y = (height + blockSize.y - 1) / blockSize.y;

	App::getLog().logInfo("Kernel (%s) block size: %d (%dx%d) | grid size: %d (%dx%d)", name, tempBlockSize, blockSize.x, blockSize.y, gridSize.x * gridSize.y, gridSize.x, gridSize.y);
}
