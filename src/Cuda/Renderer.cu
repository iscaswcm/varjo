#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <cstdint>

#include <cuda/hip/hip_vector_types.h>
#include <>

#include "Cuda/Renderer.h"
#include "Cuda/CudaUtils.h"
#include "Utils/App.h"
#include "Core/Ray.h"
#include "Core/Intersection.h"

using namespace Varjo;

namespace
{
	__device__ void initRay(Ray& ray)
	{
		ray.invD = make_float3(1.0, 1.0f, 1.0f) / ray.direction;
		ray.OoD = ray.origin / ray.direction;
	}

	__device__ Ray getRay(float2 pointOnFilm, const CameraData& camera)
	{
		float dx = pointOnFilm.x - camera.halfFilmWidth;
		float dy = pointOnFilm.y - camera.halfFilmHeight;

		float3 positionOnFilm = camera.filmCenter + (dx * camera.right) + (dy * camera.up);

		Ray ray;
		ray.origin = camera.position;
		ray.direction = normalize(positionOnFilm - camera.position);
		initRay(ray);

		return ray;
	}

	__device__ void intersectTriangle(const Triangle& triangle, const Ray& ray, Intersection& intersection)
	{
		bool hit = true;

		float3 v0v1 = triangle.vertices[1] - triangle.vertices[0];
		float3 v0v2 = triangle.vertices[2] - triangle.vertices[0];

		float3 pvec = cross(ray.direction, v0v2);
		float determinant = dot(v0v1, pvec);

		if (determinant == 0.0f)
			hit = false;

		float invDeterminant = 1.0f / determinant;

		float3 tvec = ray.origin - triangle.vertices[0];
		float u = dot(tvec, pvec) * invDeterminant;

		if (u < 0.0f || u > 1.0f)
			hit = false;

		float3 qvec = cross(tvec, v0v1);
		float v = dot(ray.direction, qvec) * invDeterminant;

		if (v < 0.0f || (u + v) > 1.0f)
			hit = false;

		float distance = dot(v0v2, qvec) * invDeterminant;

		if (distance < 0.0f || distance < ray.minDistance || distance > ray.maxDistance || distance > intersection.distance)
			hit = false;

		float w = 1.0f - u - v;

		if (hit)
		{
			intersection.wasFound = true;
			intersection.distance = distance;
			intersection.position = ray.origin + (distance * ray.direction);
			intersection.normal = w * triangle.normals[0] + u * triangle.normals[1] + v * triangle.normals[2];
			//intersection.texcoord = w * triangle.texcoords[0] + u * triangle.texcoords[1] + v * triangle.texcoords[2];
			intersection.materialIndex = triangle.materialIndex;
		}
	}

	__device__ bool intersectAabb(const AABB& aabb, const Ray& ray)
	{
		float x0 = fmaf(aabb.min.x, ray.invD.x, -ray.OoD.x);
		float y0 = fmaf(aabb.min.y, ray.invD.y, -ray.OoD.y);
		float z0 = fmaf(aabb.min.z, ray.invD.z, -ray.OoD.z);
		float x1 = fmaf(aabb.max.x, ray.invD.x, -ray.OoD.x);
		float y1 = fmaf(aabb.max.y, ray.invD.y, -ray.OoD.y);
		float z1 = fmaf(aabb.max.z, ray.invD.z, -ray.OoD.z);

		float tminbox = fmaxf(fmaxf(ray.minDistance, fminf(x0, x1)), fmaxf(fminf(y0, y1), fminf(z0, z1)));
		float tmaxbox = fminf(fminf(ray.maxDistance, fmaxf(x0, x1)), fminf(fmaxf(y0, y1), fmaxf(z0, z1)));
		
		return (tminbox <= tmaxbox);
	}

	__device__ void intersectBvh(const BVHNode* __restrict nodes, const Triangle* __restrict triangles, const Ray& ray, Intersection& intersection)
	{
		uint32_t stack[64];
		uint32_t stackIndex = 1;
		stack[0] = 0;

		BVHNode node;

		while (stackIndex > 0)
		{
			uint32_t nodeIndex = stack[--stackIndex];
			node = nodes[nodeIndex];

			// leaf node
			if (node.rightOffset == 0)
			{
				for (int i = 0; i < node.triangleCount; ++i)
					intersectTriangle(triangles[node.triangleOffset + i], ray, intersection);

				continue;
			}

			if (intersectAabb(node.aabb, ray))
			{
				stack[stackIndex++] = nodeIndex + 1; // left child
				stack[stackIndex++] = nodeIndex + uint32_t(node.rightOffset); // right child
			}
		}
	}

	__global__ void traceKernel(const CameraData* __restrict camera,
		const BVHNode* __restrict nodes,
		const Triangle* __restrict triangles,
		const Material* __restrict materials,
		hipSurfaceObject_t film, uint32_t filmWidth, uint32_t filmHeight)
	{
		uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

		Ray ray = getRay(make_float2(x, y), *camera);
		Intersection intersection;
		float3 color = make_float3(0.0f, 0.0f, 0.0f);

		intersectBvh(nodes, triangles, ray, intersection);

		if (intersection.wasFound)
			color = materials[intersection.materialIndex].baseColor * dot(ray.direction, -intersection.normal);

		surf2Dwrite(make_float4(color, 1.0f), film, x * sizeof(float4), y, hipBoundaryModeZero);
	}
}

void Renderer::initialize(const Scene& scene)
{
	CudaUtils::checkError(hipMallocManaged(&camera, sizeof(CameraData)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&nodes, sizeof(BVHNode) * scene.nodes.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&triangles, sizeof(Triangle) * scene.triangles.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&materials, sizeof(Material) * scene.materials.size()), "Could not allocate CUDA device memory");
	
	CameraData cameraData = scene.camera.getCameraData();

	memcpy(camera, &cameraData, sizeof(CameraData));
	memcpy(nodes, scene.nodes.data(), sizeof(BVHNode) * scene.nodes.size());
	memcpy(triangles, scene.triangles.data(), sizeof(Triangle) * scene.triangles.size());
	memcpy(materials, scene.materials.data(), sizeof(Material) * scene.materials.size());
}

void Renderer::shutdown()
{
	CudaUtils::checkError(hipFree(camera), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(nodes), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(triangles), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(materials), "Could not free CUDA device memory");
}

void Renderer::update(const Scene& scene)
{
	CameraData cameraData = scene.camera.getCameraData();
	memcpy(camera, &cameraData, sizeof(CameraData));
}

void Renderer::filmResized(uint32_t width, uint32_t height)
{
	CudaUtils::calculateDimensions(static_cast<void*>(traceKernel), "trace", width, height, traceKernelBlockDim, traceKernelGridDim);
}

void Renderer::render()
{
	const Film& film = App::getWindow().getFilm();
	hipGraphicsResource* filmTextureResource = film.getTextureResource();
	CudaUtils::checkError(hipGraphicsMapResources(1, &filmTextureResource, 0), "Could not map CUDA texture resource");

	hipArray_t filmTextureArray;
	CudaUtils::checkError(hipGraphicsSubResourceGetMappedArray(&filmTextureArray, filmTextureResource, 0, 0), "Could not get CUDA mapped array");

	hipResourceDesc filmResourceDesc;
	memset(&filmResourceDesc, 0, sizeof(filmResourceDesc));
	filmResourceDesc.resType = hipResourceTypeArray;
	filmResourceDesc.res.array.array = filmTextureArray;

	hipSurfaceObject_t filmSurfaceObject;
	CudaUtils::checkError(hipCreateSurfaceObject(&filmSurfaceObject, &filmResourceDesc), "Could not create CUDA surface object");

	traceKernel<<<traceKernelGridDim, traceKernelBlockDim>>>(camera, nodes, triangles, materials, filmSurfaceObject, film.getWidth(), film.getHeight());
	
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");

	CudaUtils::checkError(hipDestroySurfaceObject(filmSurfaceObject), "Could not destroy CUDA surface object");
	CudaUtils::checkError(hipGraphicsUnmapResources(1, &filmTextureResource, 0), "Could not unmap CUDA texture resource");
}
