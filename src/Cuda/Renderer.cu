#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <cstdint>

#include <cuda/hip/hip_vector_types.h>
#include <>

#include "Cuda/Renderer.h"
#include "Cuda/CudaUtils.h"
#include "Utils/App.h"
#include "Core/Ray.h"
#include "Core/Intersection.h"

using namespace Varjo;

namespace
{
	// http://www.pcg-random.org/
	__device__ uint32_t randomInt(Random& random)
	{
		uint64_t oldstate = random.state;
		random.state = oldstate * 6364136223846793005ULL + random.inc;
		uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
		uint32_t rot = oldstate >> 59u;
		return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
	}

	__device__ uint32_t randomInt(Random& random, uint32_t max)
	{
		uint32_t threshold = -max % max;

		for (;;)
		{
			uint32_t r = randomInt(random);

			if (r >= threshold)
				return r % max; // 0 <= r < max
		}
	}

	__device__ float randomFloat(Random& random)
	{
		//return float(ldexp(randomInt(random), -32)); // http://mumble.net/~campbell/tmp/random_real.c
		return float(randomInt(random)) / float(0xFFFFFFFF);
	}

	__device__ void initRandom(Random& random, uint64_t initstate, uint64_t initseq)
	{
		random.state = 0U;
		random.inc = (initseq << 1u) | 1u;
		randomInt(random);
		random.state += initstate;
		randomInt(random);
	}

	__device__ uint32_t permute(uint32_t i, uint32_t l, uint32_t p)
	{
		uint32_t w = l - 1;

		w |= w >> 1;
		w |= w >> 2;
		w |= w >> 4;
		w |= w >> 8;
		w |= w >> 16;

		do
		{
			i ^= p;
			i *= 0xe170893d;
			i ^= p >> 16;
			i ^= (i & w) >> 4;
			i ^= p >> 8;
			i *= 0x0929eb3f;
			i ^= p >> 23;
			i ^= (i & w) >> 1;
			i *= 1 | p >> 27;
			i *= 0x6935fa69;
			i ^= (i & w) >> 11;
			i *= 0x74dcb303;
			i ^= (i & w) >> 2;
			i *= 0x9e501cc3;
			i ^= (i & w) >> 2;
			i *= 0xc860a3df;
			i &= w;
			i ^= i >> 5;
		}
		while (i >= l);

		return (i + p) % l;
	}

	// http://graphics.pixar.com/library/MultiJitteredSampling/
	__device__ float2 getSample(uint32_t s, uint32_t m, uint32_t n, uint32_t p)
	{
		// if s is not permutated, the samples will come out in scanline order
		s = permute(s, m * n, p * 0xa511e9b3);

		uint32_t x = s % m;
		uint32_t y = s / m;

		uint32_t sx = permute(x, m, p * 0xa511e9b3);
		uint32_t sy = permute(y, n, p * 0x63d83595);

		float2 r;

		r.x = float(x) + float(sy) / float(n);
		r.y = float(y) + float(sx) / float(m);

		return r;
	}

	__device__ float mitchellFilter(float s)
	{
		const float B = 1.0f / 3.0f;
		const float C = 1.0f / 3.0f;

		s = abs(s);

		if (s < 1.0f)
			return ((12.0f - 9.0f * B - 6.0f * C) * (s * s * s) + (-18.0f + 12.0f * B + 6.0f * C) * (s * s) + (6.0f - 2.0f * B)) * (1.0f / 6.0f);

		if (s < 2.0f)
			return ((-B - 6.0f * C) * (s * s * s) + (6.0f * B + 30.0f * C) * (s * s) + (-12.0f * B - 48.0f * C) * s + (8.0f * B + 24.0f * C)) * (1.0f / 6.0f);

		return 0.0f;
	}

	__device__ void initRay(Ray& ray)
	{
		ray.invD = make_float3(1.0, 1.0f, 1.0f) / ray.direction;
		ray.OoD = ray.origin / ray.direction;
	}

	__device__ Ray getCameraRay(const CameraData& camera, float2 filmPoint)
	{
		float dx = filmPoint.x - camera.halfFilmWidth;
		float dy = filmPoint.y - camera.halfFilmHeight;

		float3 position = camera.filmCenter + (dx * camera.right) + (dy * camera.up);

		Ray ray;
		ray.origin = camera.position;
		ray.direction = normalize(position - camera.position);
		initRay(ray);

		return ray;
	}

	// http://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/moller-trumbore-ray-triangle-intersection
	__device__ void intersectTriangle(const Triangle* __restrict triangles, uint32_t triangleIndex, const Ray& ray, Intersection& intersection)
	{
		const Triangle triangle = triangles[triangleIndex];

		float3 v0v1 = triangle.vertices[1] - triangle.vertices[0];
		float3 v0v2 = triangle.vertices[2] - triangle.vertices[0];

		float3 pvec = cross(ray.direction, v0v2);
		float determinant = dot(v0v1, pvec);

		if (determinant == 0.0f)
			return;

		float invDeterminant = 1.0f / determinant;

		float3 tvec = ray.origin - triangle.vertices[0];
		float u = dot(tvec, pvec) * invDeterminant;

		if (u < 0.0f || u > 1.0f)
			return;

		float3 qvec = cross(tvec, v0v1);
		float v = dot(ray.direction, qvec) * invDeterminant;

		if (v < 0.0f || (u + v) > 1.0f)
			return;

		float distance = dot(v0v2, qvec) * invDeterminant;

		if (distance < 0.0f || distance < ray.minDistance || distance > ray.maxDistance || distance > intersection.distance)
			return;

		float w = 1.0f - u - v;

		intersection.wasFound = true;
		intersection.distance = distance;
		intersection.position = ray.origin + (distance * ray.direction);
		intersection.normal = w * triangle.normals[0] + u * triangle.normals[1] + v * triangle.normals[2];
		//intersection.texcoord = w * triangle.texcoords[0] + u * triangle.texcoords[1] + v * triangle.texcoords[2];
		intersection.triangleIndex = triangleIndex;
		intersection.materialIndex = triangle.materialIndex;
	}

	// https://mediatech.aalto.fi/~timo/publications/aila2012hpg_techrep.pdf
	__device__ bool intersectAabb(const AABB& aabb, const Ray& ray)
	{
		float x0 = fmaf(aabb.min.x, ray.invD.x, -ray.OoD.x);
		float y0 = fmaf(aabb.min.y, ray.invD.y, -ray.OoD.y);
		float z0 = fmaf(aabb.min.z, ray.invD.z, -ray.OoD.z);
		float x1 = fmaf(aabb.max.x, ray.invD.x, -ray.OoD.x);
		float y1 = fmaf(aabb.max.y, ray.invD.y, -ray.OoD.y);
		float z1 = fmaf(aabb.max.z, ray.invD.z, -ray.OoD.z);

		float tminbox = fmaxf(fmaxf(ray.minDistance, fminf(x0, x1)), fmaxf(fminf(y0, y1), fminf(z0, z1)));
		float tmaxbox = fminf(fminf(ray.maxDistance, fmaxf(x0, x1)), fminf(fmaxf(y0, y1), fmaxf(z0, z1)));

		return (tminbox <= tmaxbox);
	}

	__device__ void intersectBvh(const BVHNode* __restrict nodes, const Triangle* __restrict triangles, const Ray& ray, Intersection& intersection)
	{
		BVHNode node;
		uint32_t stack[64];
		uint32_t stackIndex = 1;
		stack[0] = 0;

		while (stackIndex > 0)
		{
			uint32_t nodeIndex = stack[--stackIndex];
			node = nodes[nodeIndex];

			// leaf node
			if (node.rightOffset == 0)
			{
				for (int i = 0; i < node.triangleCount; ++i)
					intersectTriangle(triangles, node.triangleOffset + i, ray, intersection);

				continue;
			}

			if (intersectAabb(node.aabb, ray))
			{
				stack[stackIndex++] = nodeIndex + 1; // left child
				stack[stackIndex++] = nodeIndex + uint32_t(node.rightOffset); // right child
			}
		}
	}

	__global__ void initPathsKernel(Path* paths, uint64_t seed, uint32_t pathCount)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= pathCount)
			return;

		initRandom(paths[id].random, seed, uint64_t(id));

		paths[id].filmSample.index = 0;
		paths[id].filmSample.permutation = randomInt(paths[id].random);
		paths[id].filmSamplePosition = make_float2(0.0f, 0.0f);
		paths[id].throughput = make_float3(0.0f, 0.0f, 0.0f);
		paths[id].result = make_float3(0.0f, 0.0f, 0.0f);
	}

	__global__ void clearPixelsKernel(Pixel* pixels, uint32_t pixelCount)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= pixelCount)
			return;

		pixels[id].color = make_float3(0.0f, 0.0f, 0.0f);
		pixels[id].weight = 0.0f;
	}

	__global__ void writePixelsKernel(Pixel* pixels, uint32_t pixelCount, hipSurfaceObject_t film, uint32_t filmWidth)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= pixelCount)
			return;

		uint32_t x = id % filmWidth;
		uint32_t y = id / filmWidth;
		float3 color = make_float3(0.0f, 0.0f, 0.0f);
		float weight = pixels[id].weight;

		if (weight != 0.0f)
			color = pixels[id].color / weight;

		const float invGamma = 1.0f / 2.2f;
		color = clamp(color, 0.0f, 1.0f);
		color.x = powf(color.x, invGamma);
		color.y = powf(color.y, invGamma);
		color.z = powf(color.z, invGamma);
		
		surf2Dwrite(make_float4(color, 1.0f), film, x * sizeof(float4), y, hipBoundaryModeZero);
	}

	// https://mediatech.aalto.fi/~samuli/publications/laine2013hpg_paper.pdf
	__global__ void logicKernel(
		Path* __restrict paths,
		Pixel* __restrict pixels,
		Queues* __restrict queues,
		uint32_t pathCount,
		uint32_t filmWidth,
		uint32_t filmHeight)
	{
		const uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= pathCount)
			return;

		Path path = paths[id];

		// add russian roulette

		if (!path.intersection.wasFound || isZero(path.throughput)) // path is terminated
		{
			int ox = int(path.filmSamplePosition.x);
			int oy = int(path.filmSamplePosition.y);

			for (int tx = -1; tx <= 2; ++tx)
			{
				for (int ty = -1; ty <= 2; ++ty)
				{
					int px = ox + tx;
					int py = oy + ty;
					px = clamp(px, 0, int(filmWidth));
					py = clamp(py, 0, int(filmHeight));
					float2 pixelPosition = make_float2(float(px), float(py));
					float2 distance = pixelPosition - path.filmSamplePosition;
					float weight = mitchellFilter(distance.x) * mitchellFilter(distance.y);
					float3 result = weight * path.result;
					int pixelIndex = py * int(filmWidth) + px;

					atomicAdd(&(pixels[pixelIndex].color.x), result.x);
					atomicAdd(&(pixels[pixelIndex].color.y), result.y);
					atomicAdd(&(pixels[pixelIndex].color.z), result.z);
					atomicAdd(&(pixels[pixelIndex].weight), weight);
				}
			}

			uint32_t queueIndex = atomicAdd(&queues->newPathQueueLength, 1);
			queues->newPathQueue[queueIndex] = id;
		}
		else // determine intersection material, add to material queue
		{
			uint32_t queueIndex = atomicAdd(&queues->materialQueueLength, 1);
			queues->materialQueue[queueIndex] = id;
		}
	}

	__global__ void newPathKernel(
		const uint32_t* __restrict newPathQueue,
		const CameraData* __restrict camera,
		Path* __restrict paths,
		Queues* __restrict queues,
		uint32_t newPathQueueLength,
		uint32_t filmWidth,
		uint32_t filmHeight,
		uint32_t filmLength)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= newPathQueueLength)
			return;

		id = newPathQueue[id];

		uint32_t filmSampleIndex = paths[id].filmSample.index;
		uint32_t filmSamplePermutation = paths[id].filmSample.permutation;

		if (filmSampleIndex >= filmLength)
		{
			filmSampleIndex = 0;
			paths[id].filmSample.permutation = ++filmSamplePermutation;
		}

		float2 filmSamplePosition = getSample(filmSampleIndex++, filmWidth, filmHeight, filmSamplePermutation);
		paths[id].filmSample.index = filmSampleIndex;
		paths[id].filmSamplePosition = filmSamplePosition;
		paths[id].throughput = make_float3(1.0f, 1.0f, 1.0f);
		paths[id].result = make_float3(0.0f, 0.0f, 0.0f);
		paths[id].ray = getCameraRay(*camera, filmSamplePosition);

		uint32_t queueIndex = atomicAdd(&queues->extensionRayQueueLength, 1);
		queues->extensionRayQueue[queueIndex] = id;
	}

	__global__ void materialKernel(
		const uint32_t* __restrict materialQueue,
		const Material* __restrict materials,
		Path* __restrict paths,
		uint32_t materialQueueLength)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= materialQueueLength)
			return;

		id = materialQueue[id];

		Intersection intersection = paths[id].intersection;

		if (intersection.wasFound)
		{
			Material material = materials[intersection.materialIndex];
			paths[id].result = material.baseColor * dot(paths[id].ray.direction, -intersection.normal);
		}
		
		paths[id].throughput = make_float3(0.0f, 0.0f, 0.0f);
	}

	__global__ void extensionRayKernel(
		const uint32_t* __restrict extensionRayQueue,
		const BVHNode* __restrict nodes,
		const Triangle* __restrict triangles,
		Path* __restrict paths,
		uint32_t extensionRayQueueLength)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= extensionRayQueueLength)
			return;

		id = extensionRayQueue[id];

		Ray ray = paths[id].ray;
		Intersection intersection;
		intersectBvh(nodes, triangles, ray, intersection);
		paths[id].intersection = intersection;
	}
}

void Renderer::initialize(const Scene& scene)
{
	CudaUtils::checkError(hipMallocManaged(&camera, sizeof(CameraData)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&nodes, sizeof(BVHNode) * scene.nodes.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&triangles, sizeof(Triangle) * scene.triangles.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&emitters, sizeof(Triangle) * scene.emitters.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&materials, sizeof(Material) * scene.materials.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths, sizeof(Path) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues, sizeof(Queues)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->newPathQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->materialQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->extensionRayQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->shadowRayQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");

	CameraData cameraData = scene.camera.getCameraData();

	memcpy(camera, &cameraData, sizeof(CameraData));
	memcpy(nodes, scene.nodes.data(), sizeof(BVHNode) * scene.nodes.size());
	memcpy(triangles, scene.triangles.data(), sizeof(Triangle) * scene.triangles.size());
	memcpy(emitters, scene.emitters.data(), sizeof(Triangle) * scene.emitters.size());
	memcpy(materials, scene.materials.data(), sizeof(Material) * scene.materials.size());

	uint64_t time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
	int blockSize, gridSize;
	CudaUtils::calculateDimensions(static_cast<void*>(initPathsKernel), "initPaths", pathCount, blockSize, gridSize);
	initPathsKernel<<<gridSize, blockSize>>>(paths, time, pathCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");

	queues->newPathQueueLength = 0;
	queues->materialQueueLength = 0;
	queues->extensionRayQueueLength = 0;
	queues->shadowRayQueueLength = 0;

	CudaUtils::calculateDimensions(static_cast<void*>(logicKernel), "logicKernel", pathCount, logicBlockSize, logicGridSize);
	CudaUtils::calculateDimensions(static_cast<void*>(newPathKernel), "newPathKernel", pathCount, newPathBlockSize, newPathGridSize);
	CudaUtils::calculateDimensions(static_cast<void*>(materialKernel), "materialKernel", pathCount, materialBlockSize, materialGridSize);
	CudaUtils::calculateDimensions(static_cast<void*>(extensionRayKernel), "extensionRayKernel", pathCount, extensionRayBlockSize, extensionRayGridSize);
	//CudaUtils::calculateDimensions(static_cast<void*>(shadowRayKernel), "shadowRayKernel", pathCount, shadowRayBlockSize, shadowRayGridSize);

	averagePathsPerSecond.setAlpha(0.05f);
	averageRaysPerSecond.setAlpha(0.05f);
}

void Renderer::shutdown()
{
	CudaUtils::checkError(hipFree(camera), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(nodes), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(triangles), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(emitters), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(materials), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->newPathQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->materialQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->extensionRayQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->shadowRayQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues), "Could not free CUDA device memory");
}

void Renderer::update(const Scene& scene)
{
	CameraData cameraData = scene.camera.getCameraData();
	memcpy(camera, &cameraData, sizeof(CameraData));
}

void Renderer::filmResized(uint32_t filmWidth, uint32_t filmHeight)
{
	if (pixels != nullptr)
		CudaUtils::checkError(hipFree(pixels), "Could not free CUDA device memory");

	pixelCount = filmWidth * filmHeight;
	CudaUtils::checkError(hipMallocManaged(&pixels, sizeof(Pixel) * pixelCount), "Could not allocate CUDA device memory");
	CudaUtils::calculateDimensions(static_cast<void*>(clearPixelsKernel), "clearPixelsKernel", pixelCount, clearPixelsBlockSize, clearPixelsGridSize);
	CudaUtils::calculateDimensions(static_cast<void*>(writePixelsKernel), "writePixelsKernel", pixelCount, writePixelsBlockSize, writePixelsGridSize);

	clear();
}

void Renderer::clear()
{
	clearPixelsKernel<<<clearPixelsGridSize, clearPixelsBlockSize>>>(pixels, pixelCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
}

void Renderer::render()
{
	Film& film = App::getWindow().getFilm();

	logicKernel<<<logicGridSize, logicBlockSize>>>(paths, pixels, queues, pathCount, film.getWidth(), film.getHeight());
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");

	if (queues->newPathQueueLength > 0)
	{
		newPathGridSize = (queues->newPathQueueLength + newPathBlockSize - 1) / newPathBlockSize;
		newPathKernel<<<newPathGridSize, newPathBlockSize>>>(queues->newPathQueue, camera, paths, queues, queues->newPathQueueLength, film.getWidth(), film.getHeight(), film.getLength());
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	}

	if (queues->materialQueueLength > 0)
	{
		materialGridSize = (queues->materialQueueLength + materialBlockSize - 1) / materialBlockSize;
		materialKernel<<<materialGridSize, materialBlockSize>>>(queues->materialQueue, materials, paths, queues->materialQueueLength);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	}

	if (queues->extensionRayQueueLength > 0)
	{
		extensionRayGridSize = (queues->extensionRayQueueLength + extensionRayBlockSize - 1) / extensionRayBlockSize;
		extensionRayKernel<<<extensionRayGridSize, extensionRayBlockSize>>>(queues->extensionRayQueue, nodes, triangles, paths, queues->extensionRayQueueLength);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	}

	/*if (queues->shadowRayQueueLength > 0)
	{
		shadowRayGridSize = (queues->shadowRayQueueLength + shadowRayBlockSize - 1) / shadowRayBlockSize;
		shadowRayKernel<<<shadowRayGridSize, shadowRayBlockSize>>>(queues->shadowRayQueue, queues->shadowRayQueueLength, paths, nodes, triangles);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	}*/

	hipSurfaceObject_t filmSurfaceObject = film.getFilmSurfaceObject();
	writePixelsKernel<<<writePixelsGridSize, writePixelsBlockSize>>>(pixels, pixelCount, filmSurfaceObject, film.getWidth());
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	film.releaseFilmSurfaceObject();

	float elapsedSeconds = timer.getElapsedSeconds();
	averagePathsPerSecond.addMeasurement(float(queues->newPathQueueLength) / elapsedSeconds);
	averageRaysPerSecond.addMeasurement(float(queues->extensionRayQueueLength + queues->shadowRayQueueLength) / elapsedSeconds);
	timer.restart();

	queues->newPathQueueLength = 0;
	queues->materialQueueLength = 0;
	queues->extensionRayQueueLength = 0;
	queues->shadowRayQueueLength = 0;
}

float Renderer::getPathsPerSecond() const
{
	return averagePathsPerSecond.getAverage();
}

float Renderer::getRaysPerSecond() const
{
	return averageRaysPerSecond.getAverage();
}
