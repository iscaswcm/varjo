#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <cstdint>

#include <cuda/hip/hip_vector_types.h>
#include <>

#include "Cuda/Renderer.h"
#include "Cuda/CudaUtils.h"
#include "Utils/App.h"
#include "Core/Ray.h"
#include "Core/Intersection.h"
#include "Core/Random.h"

using namespace Varjo;

namespace
{
	// http://www.pcg-random.org/
	__device__ uint32_t randomInt(Random& random)
	{
		uint64_t oldstate = random.state;
		random.state = oldstate * 6364136223846793005ULL + random.inc;
		uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
		uint32_t rot = oldstate >> 59u;
		return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
	}

	__device__ float randomFloat(Random& random)
	{
		//return float(ldexp(randomInt(random), -32)); // http://mumble.net/~campbell/tmp/random_real.c
		return float(randomInt(random)) / float(0xFFFFFFFF);
	}

	__device__ void initRandom(Random& random, uint64_t initstate, uint64_t initseq)
	{
		random.state = 0U;
		random.inc = (initseq << 1u) | 1u;
		randomInt(random);
		random.state += initstate;
		randomInt(random);
	}

	__device__ uint32_t permute(uint32_t i, uint32_t l, uint32_t p)
	{
		uint32_t w = l - 1;

		w |= w >> 1; w |= w >> 2;
		w |= w >> 4; w |= w >> 8;
		w |= w >> 16;

		do
		{
			i ^= p; i *= 0xe170893d;
			i ^= p >> 16; i ^= (i & w) >> 4;
			i ^= p >> 8; i *= 0x0929eb3f;
			i ^= p >> 23; i ^= (i & w) >> 1;
			i *= 1 | p >> 27; i *= 0x6935fa69;
			i ^= (i & w) >> 11; i *= 0x74dcb303;
			i ^= (i & w) >> 2; i *= 0x9e501cc3;
			i ^= (i & w) >> 2; i *= 0xc860a3df;
			i &= w; i ^= i >> 5;
		} while (i >= l);

		return (i + p) % l;
	}

	// http://graphics.pixar.com/library/MultiJitteredSampling/
	__device__ float2 getSample(uint32_t s, uint32_t m, uint32_t n, uint32_t p)
	{
		// if s is not permutated, the samples will come out in scanline order
		s = permute(s, m * n, p * 0xa511e9b3);

		uint32_t x = s % m;
		uint32_t y = s / m;

		uint32_t sx = permute(x, m, p * 0xa511e9b3);
		uint32_t sy = permute(y, n, p * 0x63d83595);

		float2 r;

		r.x = (float(x) + float(sy) / float(n)) / float(m);
		r.y = (float(y) + float(sx) / float(m)) / float(n);

		return r;
	}

	__device__ void initRay(Ray& ray)
	{
		ray.invD = make_float3(1.0, 1.0f, 1.0f) / ray.direction;
		ray.OoD = ray.origin / ray.direction;
	}

	__device__ Ray getRay(float2 pointOnFilm, const CameraData& camera)
	{
		float dx = pointOnFilm.x - camera.halfFilmWidth;
		float dy = pointOnFilm.y - camera.halfFilmHeight;

		float3 positionOnFilm = camera.filmCenter + (dx * camera.right) + (dy * camera.up);

		Ray ray;
		ray.origin = camera.position;
		ray.direction = normalize(positionOnFilm - camera.position);
		initRay(ray);

		return ray;
	}

	// http://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/moller-trumbore-ray-triangle-intersection
	__device__ void intersectTriangle(const Triangle& triangle, const Ray& ray, Intersection& intersection)
	{
		bool hit = true;

		float3 v0v1 = triangle.vertices[1] - triangle.vertices[0];
		float3 v0v2 = triangle.vertices[2] - triangle.vertices[0];

		float3 pvec = cross(ray.direction, v0v2);
		float determinant = dot(v0v1, pvec);

		if (determinant == 0.0f)
			hit = false;

		float invDeterminant = 1.0f / determinant;

		float3 tvec = ray.origin - triangle.vertices[0];
		float u = dot(tvec, pvec) * invDeterminant;

		if (u < 0.0f || u > 1.0f)
			hit = false;

		float3 qvec = cross(tvec, v0v1);
		float v = dot(ray.direction, qvec) * invDeterminant;

		if (v < 0.0f || (u + v) > 1.0f)
			hit = false;

		float distance = dot(v0v2, qvec) * invDeterminant;

		if (distance < 0.0f || distance < ray.minDistance || distance > ray.maxDistance || distance > intersection.distance)
			hit = false;

		float w = 1.0f - u - v;

		if (hit)
		{
			intersection.wasFound = true;
			intersection.distance = distance;
			intersection.position = ray.origin + (distance * ray.direction);
			intersection.normal = w * triangle.normals[0] + u * triangle.normals[1] + v * triangle.normals[2];
			//intersection.texcoord = w * triangle.texcoords[0] + u * triangle.texcoords[1] + v * triangle.texcoords[2];
			intersection.materialIndex = triangle.materialIndex;
		}
	}

	// https://mediatech.aalto.fi/~timo/publications/aila2012hpg_techrep.pdf
	__device__ bool intersectAabb(const AABB& aabb, const Ray& ray)
	{
		float x0 = fmaf(aabb.min.x, ray.invD.x, -ray.OoD.x);
		float y0 = fmaf(aabb.min.y, ray.invD.y, -ray.OoD.y);
		float z0 = fmaf(aabb.min.z, ray.invD.z, -ray.OoD.z);
		float x1 = fmaf(aabb.max.x, ray.invD.x, -ray.OoD.x);
		float y1 = fmaf(aabb.max.y, ray.invD.y, -ray.OoD.y);
		float z1 = fmaf(aabb.max.z, ray.invD.z, -ray.OoD.z);

		float tminbox = fmaxf(fmaxf(ray.minDistance, fminf(x0, x1)), fmaxf(fminf(y0, y1), fminf(z0, z1)));
		float tmaxbox = fminf(fminf(ray.maxDistance, fmaxf(x0, x1)), fminf(fmaxf(y0, y1), fmaxf(z0, z1)));

		return (tminbox <= tmaxbox);
	}

	__device__ void intersectBvh(const BVHNode* __restrict nodes, const Triangle* __restrict triangles, const Ray& ray, Intersection& intersection)
	{
		BVHNode node;
		uint32_t stack[64];
		uint32_t stackIndex = 1;
		stack[0] = 0;

		while (stackIndex > 0)
		{
			uint32_t nodeIndex = stack[--stackIndex];
			node = nodes[nodeIndex];

			// leaf node
			if (node.rightOffset == 0)
			{
				for (int i = 0; i < node.triangleCount; ++i)
					intersectTriangle(triangles[node.triangleOffset + i], ray, intersection);

				continue;
			}

			if (intersectAabb(node.aabb, ray))
			{
				stack[stackIndex++] = nodeIndex + 1; // left child
				stack[stackIndex++] = nodeIndex + uint32_t(node.rightOffset); // right child
			}
		}
	}

	__global__ void initPathsKernel(Path* paths, uint64_t seed, uint32_t pathCount)
	{
		uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

		if (id >= pathCount)
			return;

		initRandom(paths[id].random, seed, uint64_t(id));

		paths[id].filmSample.index = 0;
		paths[id].filmSample.permutation = randomInt(paths[id].random);
		paths[id].cameraSample.index = 0;
		paths[id].cameraSample.permutation = randomInt(paths[id].random);
	}

	__global__ void traceKernel(const CameraData* __restrict camera,
	                            const BVHNode* __restrict nodes,
	                            const Triangle* __restrict triangles,
	                            const Material* __restrict materials,
	                            hipSurfaceObject_t film, uint32_t filmWidth, uint32_t filmHeight)
	{
		uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

		Ray ray = getRay(make_float2(x, y), *camera);
		Intersection intersection;
		float3 color = make_float3(0.0f, 0.0f, 0.0f);

		intersectBvh(nodes, triangles, ray, intersection);

		if (intersection.wasFound)
			color = materials[intersection.materialIndex].baseColor * dot(ray.direction, -intersection.normal);

		surf2Dwrite(make_float4(color, 1.0f), film, x * sizeof(float4), y, hipBoundaryModeZero);
	}
}

void Renderer::initialize(const Scene& scene)
{
	CudaUtils::checkError(hipMallocManaged(&camera, sizeof(CameraData)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&nodes, sizeof(BVHNode) * scene.nodes.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&triangles, sizeof(Triangle) * scene.triangles.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&materials, sizeof(Material) * scene.materials.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths, sizeof(Path) * pathCount), "Could not allocate CUDA device memory");

	CameraData cameraData = scene.camera.getCameraData();

	memcpy(camera, &cameraData, sizeof(CameraData));
	memcpy(nodes, scene.nodes.data(), sizeof(BVHNode) * scene.nodes.size());
	memcpy(triangles, scene.triangles.data(), sizeof(Triangle) * scene.triangles.size());
	memcpy(materials, scene.materials.data(), sizeof(Material) * scene.materials.size());

	uint64_t time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();

	int blockSize, gridSize;
	CudaUtils::calculateDimensions(static_cast<void*>(initPathsKernel), "initPaths", pathCount, blockSize, gridSize);
	initPathsKernel<<<gridSize, blockSize>>>(paths, time, pathCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
}

void Renderer::shutdown()
{
	CudaUtils::checkError(hipFree(camera), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(nodes), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(triangles), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(materials), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths), "Could not free CUDA device memory");
}

void Renderer::update(const Scene& scene)
{
	CameraData cameraData = scene.camera.getCameraData();
	memcpy(camera, &cameraData, sizeof(CameraData));
}

void Renderer::filmResized(uint32_t width, uint32_t height)
{
	CudaUtils::calculateDimensions2D(static_cast<void*>(traceKernel), "trace", width, height, traceKernelBlockSize, traceKernelGridSize);
}

void Renderer::render()
{
	Film& film = App::getWindow().getFilm();
	hipSurfaceObject_t filmSurfaceObject = film.getFilmSurfaceObject();
	
	traceKernel<<<traceKernelGridSize, traceKernelBlockSize>>>(camera, nodes, triangles, materials, filmSurfaceObject, film.getWidth(), film.getHeight());

	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel");
	
	film.releaseFilmSurfaceObject();
}
