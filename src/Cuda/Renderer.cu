#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Utils/App.h"
#include "Utils/CudaUtils.h"

#include "Cuda/Renderer.h"
#include "Cuda/Camera.h"
#include "Cuda/Filtering.h"
#include "Cuda/Intersect.h"
#include "Cuda/Kernels.h"
#include "Cuda/Material.h"
#include "Cuda/Misc.h"
#include "Cuda/Math.h"
#include "Cuda/Random.h"
#include "Cuda/Sampling.h"
#include "Cuda/Structs.h"

using namespace Varjo;

void Renderer::initialize(const Scene& scene)
{
	CameraData cameraData = scene.camera.getCameraData();

	CudaUtils::checkError(hipMallocManaged(&camera, sizeof(CameraData)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&nodes, sizeof(BVHNode) * scene.nodes.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&triangles, sizeof(Triangle) * scene.triangles.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&emitters, sizeof(uint32_t) * scene.emitters.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&materials, sizeof(Material) * scene.materials.size()), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths, sizeof(Paths)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->random, sizeof(Random) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->filmSample, sizeof(Sample) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->filmSamplePosition, sizeof(float2) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->throughput, sizeof(float3) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->result, sizeof(float3) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->length, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->extensionRay, sizeof(Ray) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->extensionIntersection, sizeof(Intersection) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->extensionBrdf, sizeof(float3) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->extensionBrdfPdf, sizeof(float) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->extensionCosine, sizeof(float) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightRay, sizeof(Ray) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightEmittance, sizeof(float3) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightBrdf, sizeof(float3) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightBrdfPdf, sizeof(float) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightPdf, sizeof(float) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightCosine, sizeof(float) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&paths->lightRayBlocked, sizeof(bool) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues, sizeof(Queues)), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->newPathQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->materialQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->extensionRayQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");
	CudaUtils::checkError(hipMallocManaged(&queues->lightRayQueue, sizeof(uint32_t) * pathCount), "Could not allocate CUDA device memory");

	memcpy(camera, &cameraData, sizeof(CameraData));
	memcpy(nodes, scene.nodes.data(), sizeof(BVHNode) * scene.nodes.size());
	memcpy(triangles, scene.triangles.data(), sizeof(Triangle) * scene.triangles.size());
	memcpy(emitters, scene.emitters.data(), sizeof(uint32_t) * scene.emitters.size());
	memcpy(materials, scene.materials.data(), sizeof(Material) * scene.materials.size());

	calculateDimensions(reinterpret_cast<void*>(initPathsKernel), "initPathsKernel", pathCount, initPathsBlockSize, initPathsGridSize);
	calculateDimensions(reinterpret_cast<void*>(clearPathsKernel), "clearPathsKernel", pathCount, clearPathsBlockSize, clearPathsGridSize);
	calculateDimensions(reinterpret_cast<void*>(logicKernel), "logicKernel", pathCount, logicBlockSize, logicGridSize);
	calculateDimensions(reinterpret_cast<void*>(newPathKernel), "newPathKernel", pathCount, newPathBlockSize, newPathGridSize);
	calculateDimensions(reinterpret_cast<void*>(materialKernel), "materialKernel", pathCount, materialBlockSize, materialGridSize);
	calculateDimensions(reinterpret_cast<void*>(extensionRayKernel), "extensionRayKernel", pathCount, extensionRayBlockSize, extensionRayGridSize);
	calculateDimensions(reinterpret_cast<void*>(lightRayKernel), "lightRayKernel", pathCount, lightRayBlockSize, lightRayGridSize);

	uint64_t time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
	initPathsKernel<<<initPathsGridSize, initPathsBlockSize>>>(paths, time, pathCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel (initPaths)");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel (initPaths)");

	averagePathsPerSecond.setAlpha(0.05f);
	averageRaysPerSecond.setAlpha(0.05f);
	emitterCount = uint32_t(scene.emitters.size());

	queues->newPathQueueLength = 0;
	queues->materialQueueLength = 0;
	queues->extensionRayQueueLength = 0;
	queues->lightRayQueueLength = 0;
}

void Renderer::shutdown()
{
	CudaUtils::checkError(hipFree(camera), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(nodes), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(triangles), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(emitters), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(materials), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->random), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->filmSample), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->filmSamplePosition), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->throughput), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->result), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->length), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->extensionRay), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->extensionIntersection), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->extensionBrdf), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->extensionBrdfPdf), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->extensionCosine), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightRay), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightEmittance), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightBrdf), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightBrdfPdf), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightPdf), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightCosine), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths->lightRayBlocked), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(paths), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->newPathQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->materialQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->extensionRayQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues->lightRayQueue), "Could not free CUDA device memory");
	CudaUtils::checkError(hipFree(queues), "Could not free CUDA device memory");
}

void Renderer::update(const Scene& scene)
{
	CameraData cameraData = scene.camera.getCameraData();
	memcpy(camera, &cameraData, sizeof(CameraData));
}

void Renderer::filmResized(uint32_t filmWidth, uint32_t filmHeight)
{
	if (pixels != nullptr)
		CudaUtils::checkError(hipFree(pixels), "Could not free CUDA device memory");

	pixelCount = filmWidth * filmHeight;
	CudaUtils::checkError(hipMallocManaged(&pixels, sizeof(Pixel) * pixelCount), "Could not allocate CUDA device memory");
	calculateDimensions(reinterpret_cast<void*>(clearPixelsKernel), "clearPixelsKernel", pixelCount, clearPixelsBlockSize, clearPixelsGridSize);
	calculateDimensions(reinterpret_cast<void*>(writePixelsToFilmKernel), "writePixelsToFilmKernel", pixelCount, writePixelsToFilmBlockSize, writePixelsToFilmGridSize);

	clear();
}

void Renderer::clear()
{
	clearPathsKernel<<<clearPathsGridSize, clearPathsBlockSize>>>(paths, pathCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel (clearPaths)");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel (clearPaths)");

	clearPixelsKernel<<<clearPixelsGridSize, clearPixelsBlockSize>>>(pixels, pixelCount);
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel (clearPixels)");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel (clearPixels)");
}

void Renderer::render()
{
	Film& film = App::getWindow().getFilm();

	logicKernel<<<logicGridSize, logicBlockSize>>>(paths, queues, triangles, emitters, materials, pixels, pathCount, emitterCount, film.getWidth(), film.getHeight());
	newPathKernel<<<newPathGridSize, newPathBlockSize>>>(paths, queues, camera, film.getWidth(), film.getHeight(), film.getLength());
	materialKernel<<<materialGridSize, materialBlockSize>>>(paths, queues, materials);
	extensionRayKernel << <extensionRayGridSize, extensionRayBlockSize >> >(paths, queues, nodes, triangles);
	lightRayKernel<<<lightRayGridSize, lightRayBlockSize>>>(paths, queues, nodes, triangles);

	hipSurfaceObject_t filmSurfaceObject = film.getFilmSurfaceObject();
	writePixelsToFilmKernel<<<writePixelsToFilmGridSize, writePixelsToFilmBlockSize>>>(pixels, pixelCount, filmSurfaceObject, film.getWidth());
	CudaUtils::checkError(hipPeekAtLastError(), "Could not launch CUDA kernel (writePixels)");
	CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute CUDA kernel (writePixels)");
	film.releaseFilmSurfaceObject();

	float elapsedSeconds = timer.getElapsedSeconds();
	averagePathsPerSecond.addMeasurement(float(queues->newPathQueueLength) / elapsedSeconds);
	averageRaysPerSecond.addMeasurement(float(queues->extensionRayQueueLength + queues->lightRayQueueLength) / elapsedSeconds);
	timer.restart();

	queues->newPathQueueLength = 0;
	queues->materialQueueLength = 0;
	queues->extensionRayQueueLength = 0;
	queues->lightRayQueueLength = 0;
}

float Renderer::getPathsPerSecond() const
{
	return averagePathsPerSecond.getAverage();
}

float Renderer::getRaysPerSecond() const
{
	return averageRaysPerSecond.getAverage();
}
